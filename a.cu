
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void vectorAdd(const float *A, const float *B, float *C) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

int main(int argc, char **argv) {
  if (argc > 1) {
    printf("Received arguments: ");
  }
  for (int i = 1; i < argc; ++i) {
    printf("%s ", argv[i]);
  }
  printf("\n");

  float A[N], B[N], C[N];

  for (int i = 0; i < N; i++) {
    A[i] = i * 123.0f;
    B[i] = i * 2.0f;
  }

  float *d_A, *d_B, *d_C;

  hipMalloc((void **)&d_A, N * sizeof(float));
  hipMalloc((void **)&d_B, N * sizeof(float));
  hipMalloc((void **)&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C);
  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

  printf("Result: \n");
  for (int i = 0; i < 10; i++) {
    printf("%f + %f = %f\n", A[i], B[i], C[i]);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}

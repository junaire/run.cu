#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error in file " << __FILE__ << " at line " << __LINE__
              << ": " << hipGetErrorString(err) << std::endl;
    exit(EXIT_FAILURE);
  }
}

void checkCublasError(hipblasStatus_t stat) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "cuBLAS error in file " << __FILE__ << " at line " << __LINE__
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char *argv[]) {
  int M = 1024;
  int K = 512;
  int N = 256;

  if (argc == 4) {
    M = std::atoi(argv[1]);
    K = std::atoi(argv[2]);
    N = std::atoi(argv[3]);
  } else {
    std::cout << "Using default matrix dimensions: " << M << " x " << K << " x "
              << N << std::endl;
  }

  float *h_A = (float *)malloc(M * K * sizeof(float));
  float *h_B = (float *)malloc(K * N * sizeof(float));
  float *h_C = (float *)malloc(M * N * sizeof(float));

  for (int i = 0; i < M * K; ++i) {
    h_A[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < K * N; ++i) {
    h_B[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  float *d_A, *d_B, *d_C;

  checkCudaError(hipMalloc((void **)&d_A, M * K * sizeof(float)));
  checkCudaError(hipMalloc((void **)&d_B, K * N * sizeof(float)));
  checkCudaError(hipMalloc((void **)&d_C, M * N * sizeof(float)));

  checkCudaError(
      hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
  checkCudaError(
      hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  checkCublasError(hipblasCreate(&handle));

  const float alpha = 1.0f;
  const float beta = 0.0f;

  hipEvent_t start, stop;
  checkCudaError(hipEventCreate(&start));
  checkCudaError(hipEventCreate(&stop));

  checkCudaError(hipEventRecord(start));
  checkCublasError(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
                               &alpha, d_A, M, d_B, K, &beta, d_C, M));
  checkCudaError(hipEventRecord(stop));

  checkCudaError(hipEventSynchronize(stop));

  float milliseconds = 0;
  checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));

  checkCudaError(
      hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

  std::cout << "Result C[0][0]: " << h_C[0] << std::endl;
  std::cout << "Elapsed time: " << milliseconds << " ms" << std::endl;

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  checkCublasError(hipblasDestroy(handle));

  checkCudaError(hipEventDestroy(start));
  checkCudaError(hipEventDestroy(stop));

  return 0;
}
